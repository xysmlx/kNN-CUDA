#include "hip/hip_runtime.h"
// #pragma comment(linker, "/STACK:102400000,102400000")
#include <cstdio>
#include <iostream>
#include <cstring>
#include <string>
#include <cmath>
#include <set>
#include <list>
#include <map>
#include <iterator>
#include <cstdlib>
#include <vector>
#include <queue>
#include <ctime>
#include <stack>
#include <algorithm>
#include <functional>
#include <ctime>
#include <fstream>
using namespace std;
typedef long long ll;
const double INF = 1e30;
typedef pair<int, double> prid;

class KNN
{
public:
	static const int maxCol = 10;
	static const int maxRow = 46010;

	static const int testNum = 300; // Data number for test

public:
	struct Node
	{
		double data[maxCol];
		string label;
	};

	struct CMP
	{
		bool operator ()(const prid &A, const prid &B) const
		{
			return A.second < B.second;
		}
	};

public:
	void init(int _k, int _row, int _col, string path); // init
	void input(); // Input data
	void ZScoreNorm(); // Z-Score Norm
	void MaxMinNorm(); // Max min Norm
	double dis(const Node &A, const Node &B); // Calculate distance between A and B
	void CalDis();
	string MaxFreqLabel(); // Calculate freq label
	void knn(); // Run knn

	void CUDAInit(); // Init CUDA

	void debug(); // For debug

public:
	ifstream fin;
	string filepath;

	int therK;
	int row;
	int col;

	Node dataSet[maxRow];
	Node dataTest;

	vector<prid> vec;
	map<string, int> mp;

	// Data for CUDA
	double trainmtx[maxRow][maxCol];
	int trainRow;
	double *GTrainData;
	size_t pitch_d;
	size_t pitch_h;
};

void KNN::init(int _k, int _row, int _col, string path)
{
	therK = _k;
	row = _row;
	col = _col;
	filepath = path;

	cout << therK << " " << row << " " << col << endl;
}

void KNN::input()
{
	fin.open(filepath);
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
			fin >> dataSet[i].data[j];
		fin >> dataSet[i].label;
	}
	fin.close();
	// debug();

	MaxMinNorm();
	// ZScoreNorm();
}

void KNN::ZScoreNorm()
{
	for (int j = 0; j < col; j++)
	{
		double avg = 0;
		for (int i = 0; i < row; i++)
			avg += dataSet[i].data[j];
		avg /= (double)row;

		double sig = 0;
		for (int i = 0; i < row; i++)
			sig += (dataSet[i].data[j] - avg) * (dataSet[i].data[j] - avg);
		sig /= (double)row;
		sig = sqrt(sig);

		for (int i = 0; i < row; i++)
			dataSet[i].data[j] = (dataSet[i].data[j] - avg) / sig;
	}
}

void KNN::MaxMinNorm()
{
	for (int j = 0; j < col; j++)
	{
		double maxx = max(dataSet[0].data[j], dataSet[1].data[j]);
		double minx = min(dataSet[0].data[j], dataSet[1].data[j]);
		for (int i = 0; i < row; i++)
		{
			//if (dataSet[i].data[j] > maxx) maxx = dataSet[i].data[j];
			//else if (dataSet[i].data[j] < minx) minx = dataSet[i].data[j];
			maxx = max(maxx, dataSet[i].data[j]);
			minx = min(minx, dataSet[i].data[j]);
		}
		for (int i = 0; i < row; i++)
			dataSet[i].data[j] = (double)(dataSet[i].data[j] - minx) / (double)(maxx - minx);
	}
}

double KNN::dis(const Node &A, const Node &B)
{
	double ret = 0;
	for (int i = 0; i < col; i++)
		ret += (A.data[i] - B.data[i]) * (A.data[i] - B.data[i]);
	return sqrt(ret);
}

__global__ void DisKernel(double *traindata, double *testdata, double *dis, int pitch, int N, int D)
{
	int tid = blockIdx.x;
	if (tid < N)
	{
		double temp = 0;
		double sum = 0;
		for (int i = 0; i < D; i++)
		{
			temp = *((double*)((char*)traindata + tid * pitch) + i) - testdata[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}

void KNN::CalDis()
{
	vec.clear();

	double *GTestData;
	double *GDis;
	double *distance;

	distance = new double[trainRow];

	hipMalloc(&GTestData, col * sizeof(double));
	hipMalloc(&GDis, trainRow * sizeof(double));

	hipMemset(GTestData, 0, col * sizeof(double));
	hipMemset(GDis, 0, trainRow * sizeof(double));

	hipMemcpy(GTestData, dataTest.data, col * sizeof(double), hipMemcpyHostToDevice);

	DisKernel << <trainRow, 1 >> >(GTrainData, GTestData, GDis, pitch_d, trainRow, col);

	hipMemcpy(distance, GDis, trainRow * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = testNum; i < row; i++)
		vec.push_back(make_pair(i, distance[i - testNum]));

}

string KNN::MaxFreqLabel()
{
	CalDis();
	sort(vec.begin(), vec.end(), CMP());

	mp.clear();
	for (int i = 0; i < therK; i++)
		mp[dataSet[vec[i].first].label]++;

	string ret;
	int cnt = 0;
	for (auto ite = mp.begin(); ite != mp.end(); ite++)
	{
		if (ite->second > cnt)
		{
			cnt = ite->second;
			ret = ite->first;
		}
	}

	return ret;
}

void KNN::knn()
{
	cout << "Test data num: " << testNum << endl;

	int cnt = 0;
	for (int i = 0; i < testNum; i++)
	{
		dataTest = dataSet[i];
		if (MaxFreqLabel() == dataTest.label)
			cnt++;
	}

	//cout << cnt << " " << testNum << endl;

	cout << "Accuracy Rate: " << (double)cnt / (double)testNum << endl;
}

void KNN::CUDAInit()
{
	trainRow = row - testNum;
	for (int i = testNum; i < row; i++)
	{
		for (int j = 0; j < col; j++)
			trainmtx[i - testNum][j] = dataSet[i].data[j];
	}

	pitch_h = col*sizeof(double);

	hipMallocPitch(&GTrainData, &pitch_d, col * sizeof(double), trainRow);
	hipMemset(GTrainData, 0, trainRow * col * sizeof(double));
	hipMemcpy2D(GTrainData, pitch_d, trainmtx, pitch_h, col * sizeof(double), trainRow, hipMemcpyHostToDevice);
}

void KNN::debug()
{
	ofstream fout;
	fout.open("debug.txt");
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
			fout << dataSet[i].data[j] << "\t";
		fout << dataSet[i].label << endl;
	}
}

KNN knn;

void init()
{
	knn.init(7, 3000, 8, "allTypeC.txt");
}
void input()
{
	knn.input();
}
void debug()
{
	//
}
void solve()
{
	clock_t st, ed;
	st = clock();
	knn.knn();
	ed = clock();
	cout << "Time: " << ed - st << endl;
}
void output()
{
	//
}
int main()
{
	init();
	input();
	solve();
	output();

	return 0;
}